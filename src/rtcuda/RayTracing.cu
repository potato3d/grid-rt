#include "hip/hip_runtime.h"
// Relevant includes
#include <rtc/common.h>
#include <rtc/CudaDependencies.h>
#include "KernelDependencies.h"

// Include kernel
#include "RayTracing_kernel.cu"

// Forward declaration
#include "RayTracing.h"

#include <cudpp.h>

////////////////////////////////////////////////////////////////////////////////
// Globals
static GLuint g_frameBufferId = 0;

// Texture data
static int2*   g_texCellPointersData = NULL;
static int*    g_texCellTriangleIdsData = NULL;
static float4* g_texVerticesData = NULL;
static float4* g_texNormalsData = NULL;

// Camera
static Camera g_camera;

// Acceleration structure data
static Grid g_grid;

// TODO: 
//static int    g_vertexCount = 0;
//static float* g_triangleCounters = NULL;
//static float* g_triangleIndices = NULL;
//
//static CUDPPScanConfig g_scanConfig;


////////////////////////////////////////////////////////////////////////////////
// Entry point for Cuda functionality on host side
////////////////////////////////////////////////////////////////////////////////
extern "C" {

bool rtInitContext()
{
	int dummyArgc = 0;
	const char** dummyArgv = NULL;

	// Initialize CUDA context
	CUT_DEVICE_INIT( dummyArgc, dummyArgv );

	// Everything ok
	return true;
}

void cudaRegisterBuffer( GLuint bufferId )
{
	if( g_frameBufferId != 0 )
		cudaUnregisterBuffer( bufferId );

	CUDA_SAFE_CALL( cudaGLRegisterBufferObject( bufferId ) );
	CUT_CHECK_ERROR( "cudaGLRegisterBufferObject failed.\n" );
	g_frameBufferId = bufferId;
}

void cudaUnregisterBuffer( GLuint bufferId )
{
	if( bufferId != g_frameBufferId )
		return;

	CUDA_SAFE_CALL( cudaGLUnregisterBufferObject( bufferId ) );
	CUT_CHECK_ERROR( "cudaGLUnregisterBufferObject failed.\n" );
	g_frameBufferId = 0;
}

void cudaSetScreenSize( uint32 width, uint32 height )
{
	g_camera.screenWidth = width;
	g_camera.screenHeight = height;

	g_camera.invScreenWidth = 1.0f / (float)width;
	g_camera.invScreenHeight = 1.0f / (float)height;
}

void cudaSetCameraParameters( float posX, float posY, float posZ, 
							  float baseDirX, float baseDirY, float baseDirZ,
							  float uNearX, float uNearY, float uNearZ, 
							  float vNearX, float vNearY, float vNearZ )
{
	g_camera.position.x = posX;
	g_camera.position.y = posY;
	g_camera.position.z = posZ;

	g_camera.baseDir.x = baseDirX;
	g_camera.baseDir.y = baseDirY;
	g_camera.baseDir.z = baseDirZ;

	g_camera.nearU.x = uNearX;
	g_camera.nearU.y = uNearY;
	g_camera.nearU.z = uNearZ;

	g_camera.nearV.x = vNearX;
	g_camera.nearV.y = vNearY;
	g_camera.nearV.z = vNearZ;
}

void cudaTransferTriangleVertices( float* vertices, uint32 size )
{
	//g_vertexCount = size / 4;

	uint32 byteTotal = size*sizeof(float);

	// Malloc linear memory
	CUDA_SAFE_CALL( hipMalloc( (void**)&g_texVerticesData, byteTotal ) );
	CUT_CHECK_ERROR( "hipMalloc failed.\n" );

	// Memcpy to linear memory
	CUDA_SAFE_CALL( hipMemcpy( g_texVerticesData, vertices, byteTotal, hipMemcpyHostToDevice ) );
	CUT_CHECK_ERROR( "hipMemcpy failed.\n" );

	// Bind the data to the texture
    CUDA_SAFE_CALL( hipBindTexture( 0, texVertices, g_texVerticesData, byteTotal ) );
}

void cudaTransferTriangleNormals( float* normals, uint32 size )
{
	uint32 byteTotal = size*sizeof(float);

	// Malloc linear memory
	CUDA_SAFE_CALL( hipMalloc( (void**)&g_texNormalsData, byteTotal ) );
	CUT_CHECK_ERROR( "hipMalloc failed.\n" );

	// Memcpy to linear memory
	CUDA_SAFE_CALL( hipMemcpy( g_texNormalsData, normals, byteTotal, hipMemcpyHostToDevice ) );
	CUT_CHECK_ERROR( "hipMemcpy failed.\n" );

	// Bind the data to the texture
    CUDA_SAFE_CALL( hipBindTexture( 0, texNormals, g_texNormalsData, byteTotal ) );
}

void cudaTransferCellPointers( int32* cellPointers, uint32 size )
{
	uint32 byteTotal = size*sizeof(int32);

	// Malloc linear memory
	CUDA_SAFE_CALL( hipMalloc( (void**)&g_texCellPointersData, byteTotal ) );
	CUT_CHECK_ERROR( "hipMalloc failed.\n" );

	// Memcpy to linear memory
	CUDA_SAFE_CALL( hipMemcpy( g_texCellPointersData, cellPointers, byteTotal, hipMemcpyHostToDevice ) );
	CUT_CHECK_ERROR( "hipMemcpy failed.\n" );

	// Bind the data to the texture
    CUDA_SAFE_CALL( hipBindTexture( 0, texCellPointers, g_texCellPointersData, byteTotal ) );
}

void cudaTransferCellTriangleIds( int32* cellTriangleIds, uint32 size )
{
	uint32 byteTotal = size*sizeof(int32);

	// Malloc linear memory
	CUDA_SAFE_CALL( hipMalloc( (void**)&g_texCellTriangleIdsData, byteTotal ) );
	CUT_CHECK_ERROR( "hipMalloc failed.\n" );

	// Memcpy to linear memory
	CUDA_SAFE_CALL( hipMemcpy( g_texCellTriangleIdsData, cellTriangleIds, byteTotal, hipMemcpyHostToDevice ) );
	CUT_CHECK_ERROR( "hipMemcpy failed.\n" );

	// Bind the data to the texture
    CUDA_SAFE_CALL( hipBindTexture( 0, texCellTriangleIds, g_texCellTriangleIdsData, byteTotal ) );
}

void cudaSetBoxMin( float x, float y, float z )
{
	g_grid.boxMin.x = x;
	g_grid.boxMin.y = y;
	g_grid.boxMin.z = z;
}

void cudaSetBoxMax( float x, float y, float z )
{
	g_grid.boxMax.x = x;
	g_grid.boxMax.y = y;
	g_grid.boxMax.z = z;
}

void cudaSetGridParameters( float cellSizeX, float cellSizeY, float cellSizeZ,
						    int32 numCellsX, int32 numCellsY, int32 numCellsZ )
{
	g_grid.cellSize.x = cellSizeX;
	g_grid.cellSize.y = cellSizeY;
	g_grid.cellSize.z = cellSizeZ;

	g_grid.invCellSize.x = 1.0f / cellSizeX;
	g_grid.invCellSize.y = 1.0f / cellSizeY;
	g_grid.invCellSize.z = 1.0f / cellSizeZ;

	g_grid.gridSize.x = numCellsX;
	g_grid.gridSize.y = numCellsY;
	g_grid.gridSize.z = numCellsZ;

	// TODO: 
	//unsigned int numElements = numCellsX*numCellsY*numCellsZ;
	//unsigned int memSize = numElements * sizeof(float);
	//CUDA_SAFE_CALL( hipMalloc( (void**)&g_triangleCounters, memSize ) );
	//CUT_CHECK_ERROR( "hipMalloc failed.\n" );

	//CUDA_SAFE_CALL( hipMalloc( (void**)&g_triangleIndices, memSize ) );
	//CUT_CHECK_ERROR( "hipMalloc failed.\n" );

 //   g_scanConfig.direction      = CUDPP_SCAN_FORWARD;
 //   g_scanConfig.exclusivity    = CUDPP_SCAN_EXCLUSIVE;
 //   g_scanConfig.op		        = CUDPP_ADD;
 //   g_scanConfig.datatype       = CUDPP_FLOAT;
 //   g_scanConfig.maxNumElements = numElements;
 //   g_scanConfig.maxNumRows	    = 1;
 //   g_scanConfig.rowPitch       = 0;

 //   cudppInitializeScan( &g_scanConfig );
}

void cudaSetBlockSize( uint32 width, uint32 height )
{
	//g_blockWidth = width;
	//g_blockHeight = height;
}

void cudaRayTrace()
{
	// Map PBO to CUDA
	CUDA_SAFE_CALL( cudaGLMapBufferObject( (void**)&g_camera.frameBuffer, g_frameBufferId ) );

	// Setup execution configuration
	dim3 blockSize( g_blockWidth, g_blockHeight );
	dim3 gridSize( g_camera.screenWidth/g_blockWidth, g_camera.screenHeight/g_blockHeight, 1 );

	//blockSize.x = g_blockWidth;
	//blockSize.y = g_blockHeight;

	//int cellTotal = g_grid.gridSize.x * g_grid.gridSize.y * g_grid.gridSize.z;
	//gridSize.x = ceilf( cellTotal / (float)blockSize.x );

	// Call kernel
	//countTriangles<<<gridSize, blockSize>>>( g_vertexCount, g_grid, g_triangleCounters );
	//cudppScan( g_triangleIndices, g_triangleCounters, cellTotal, &g_scanConfig );

	rayTrace<<<gridSize, blockSize>>>( g_camera, g_grid );

	// Check for launch failure
	CUT_CHECK_ERROR( "Ray tracing kernel execution failed.\n" );

	// Unmap PBO back to OpenGL
	CUDA_SAFE_CALL( cudaGLUnmapBufferObject( g_frameBufferId ) );
}

} // extern "C"
