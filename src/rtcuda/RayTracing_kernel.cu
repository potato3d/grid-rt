#include "hip/hip_runtime.h"
// Kernel dependencies
#include "KernelDependencies.h"

// Global constants
static __device__ const float EPSILON   = 2e-6f;
static __device__ const float MAX_VALUE = 1e20f;

// Grid data
texture<int2, 1> texCellPointers;
texture<int, 1>  texCellTriangleIds;

// Geometry data
texture<float4, 1>  texVertices;
texture<float4, 1>  texNormals;

// 3-component texture fetch
static inline __device__ float3 tex1Dfetch3( texture<float4, 1, hipReadModeElementType> texRef, int idx )
{
	float4 texel = tex1Dfetch( texRef, idx );
	return make_float3( texel.x, texel.y, texel.z );
}

// Color conversion functions
static inline __device__ uchar4 make_color( unsigned char r, unsigned char g, unsigned char b )
{
    return make_uchar4( r, g, b, 255 );
}

static inline __device__ uchar4 make_color( unsigned char value )
{
    return make_uchar4( value, value, value, 255 );
}

static inline __device__ uchar4 make_color( float3 color, float alpha )
{
    return make_uchar4( color.x * 255.0f, color.y * 255.0f, color.z * 255.0f, alpha * 255.0f );
}

// Clip ray against bounding box
static inline __device__ bool rayHitsBoundingBox( float3 boxMin, float3 boxMax, Ray& ray )
{
	float3 t1 = ( boxMin - ray.orig ) * ray.invDir;
	float3 t2 = ( boxMax - ray.orig ) * ray.invDir;
	
	float3 minT1T2 = min( t1, t2 );
	float3 maxT1T2 = max( t1, t2 );
	
	ray.tnear = max( max( minT1T2.x, minT1T2.y ), minT1T2.z );
	ray.tfar = min( min( maxT1T2.x, maxT1T2.y ), maxT1T2.z );

	return ray.tnear <= ray.tfar;
}

// Compute intersection between triangle and current ray
// Moller-Trumbore algorithm
static inline __device__ void hitMT( int triangleId, const Ray& ray, Hit& hit )
{
	float3 v0 = tex1Dfetch3( texVertices, triangleId );
	float3 v1 = tex1Dfetch3( texVertices, triangleId + 1 );
	float3 v2 = tex1Dfetch3( texVertices, triangleId + 2 );

	float3 e1 = v1 - v0;
	float3 e2 = v2 - v0;
	float3 tvec = ray.orig - v0;

	float3 p = cross(ray.dir, e2);
	float3 q = cross(tvec, e1);

	float invdet = 1.0f / dot(p, e1);

	float u = dot(p, tvec) * invdet;
	float v = dot(q, ray.dir) * invdet;

	// Update hit
	bool isHit = (u >= 0.0f) && (v >= 0.0f) && (u + v <= 1.0f);
	
	float t = dot(q, e2) * invdet;

	// Update hit
	isHit &= (t > 0.0f) && (t < ray.tfar + EPSILON) && (t < hit.dist);

	if( isHit )
	{
		hit.id = triangleId;
		hit.u = u;
		hit.v = v;
		hit.dist = t;
	}
}

// Compute interpolated shading normal
static inline __device__ float3 computeShadingNormal( const Hit& hit )
{
	// Get triangle normals
    float3 n0 = tex1Dfetch3( texNormals, hit.id );
    float3 n1 = tex1Dfetch3( texNormals, hit.id + 1 );
    float3 n2 = tex1Dfetch3( texNormals, hit.id + 2 );

	return normalize( n0 * ( 1.0f - ( hit.u + hit.v ) ) +	// v0 coord
		              n1 * hit.u +							// v1 coord
					  n2 * hit.v );							// v2 coord
}

static inline __device__ uchar4 shade( const Ray& ray, const Hit& hit )
{
	// Need normalized ray direction
	float3 rayDirNormalized = normalize( ray.dir );

	// Compute interpolated shading normal
	float3 sampleNormal = computeShadingNormal( hit );

	// Hard-coded material information
	const float3 ambient = make_float3( 0.1f, 0.1f, 0.1f );
	const float3 diffuse = make_float3( 0.0f, 0.0f, 1.0f );

	// Headlight illumination
	float nDotD = -dot( sampleNormal, rayDirNormalized );
	float3 sampleColor = ( ambient + ( diffuse - ambient ) * nDotD ) * diffuse;
	return make_color( sampleColor, 1.0f );
}

static inline __device__ bool isEmpty( int2 cell )
{
	return cell.y == 0;
}

static inline __device__ int getTriangleStart( int2 cell )
{
	return cell.x;
}

static inline __device__ int getTriangleCount( int2 cell )
{
	return cell.y;
}

// Kernel for entire ray tracing pipeline
// Each thread traces a single ray
// Each thread computes its own ray direction
__global__ void rayTrace( Camera camera, Grid grid )
{
	//////////////////////////////////////////////////////////////////////////
	// Step 1: compute my screen coordinates
	//////////////////////////////////////////////////////////////////////////
	unsigned int screenX = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int screenY = blockIdx.y * blockDim.y + threadIdx.y;

	float uStep = (float)screenX * camera.invScreenWidth;
	float vStep = (float)screenY * camera.invScreenHeight;

	unsigned int pixelAddress = screenY*camera.screenWidth + screenX;

	//////////////////////////////////////////////////////////////////////////
	// Step 2.1: minimal ray attributes for box clipping
	//////////////////////////////////////////////////////////////////////////
	Ray ray;

	ray.orig = camera.position;

	ray.dir = camera.baseDir + camera.nearU*uStep + camera.nearV*vStep;
	ray.invDir = rcp( ray.dir );

    // If don't hit bbox in local space, no need to trace underlying triangles
	if( !rayHitsBoundingBox( grid.boxMin, grid.boxMax, ray ) )
	{
		// Background color
		camera.frameBuffer[pixelAddress] = make_color( 255 );
		return;
	}

	//////////////////////////////////////////////////////////////////////////
	// Step 2.2: remaining ray attributes
	//////////////////////////////////////////////////////////////////////////
	float3 dirSignBits;
    dirSignBits.x = ( ray.dir.x < 0.0f )? 1.0f : 0.0f;
    dirSignBits.y = ( ray.dir.y < 0.0f )? 1.0f : 0.0f;
    dirSignBits.z = ( ray.dir.z < 0.0f )? 1.0f : 0.0f;

	float3 notDirSignBits = not( dirSignBits );

	/************************************************************************/
	/* Initial setup                                                        */
	/************************************************************************/
	// 1. Find initial cell where ray begins

	// Since ray was already clipped against bbox (grid),
	// ray tnear gives us the starting t (thus the start point as well)
	float3 startPoint = ray.orig + ray.dir * ray.tnear;
	
	// Floor is needed when working with float values (equivalent to truncating to int)
	float3 cellCoords = floor( clamp( grid.worldToVoxel( startPoint ), 
		                              make_float3( 0.0f ), grid.gridSize - make_float3( 1.0f ) ) );

	// 2. Compute stepX, stepY, stepZ
	float3 cellStep = -dirSignBits + notDirSignBits;

	// 3 Compute out of grid limits
	float3 outLimit = -dirSignBits + grid.gridSize * notDirSignBits;

	// 4. Compute tDeltaX, tDeltaY, tDeltaZ
	float3 tDelta = abs( grid.cellSize * ray.invDir );

	// 5. Compute tNextX, tNextY, tNextZ
	float3 tMax = ( grid.voxelToWorld( cellCoords + notDirSignBits ) - ray.orig ) * ray.invDir;

	/************************************************************************/
	/* Trace ray through grid                                               */
	/************************************************************************/
	// Find first non-empty cell
	int2 cell = tex1Dfetch( texCellPointers, grid.to1dCoord( cellCoords ) );

	// Store hit information
	Hit hit;
	hit.dist = MAX_VALUE;

	// Minimum tMax in all 3 dimensions, used for logical comparison to determine next cell
	float minTmax;

	// Stores 1 for next cell dimension and 0 for the others, used to select next cell
	float3 comp;

	// While inside grid
	do
	{
		// Early traversal pre-computation
		// Already begin computing next cell to be visited before testing current one
		// To go to next cell, need to decide which dimension is next
		// comp stores 1 for next dimension and 0 for others
 		minTmax = min( min( tMax.x, tMax.y ), tMax.z );
 		comp = step( tMax - make_float3( minTmax ), make_float3( EPSILON ) );

		// Step ray according to comp
 		cellCoords += cellStep * comp;
 		tMax += tDelta * comp;

		// If cell contains triangles, test intersection
		if( !isEmpty( cell ) )
		{
			// We send the smallest tMax as the maximum valid distance
			// This avoids false intersections outside current cell
			ray.tfar = minTmax;

			// Iterate through triangles in given cell and compute nearest intersection, if any
			int i = getTriangleStart( cell );
			int end = getTriangleStart( cell ) + getTriangleCount( cell );

			while( i < end )
			{
				// Get triangle id
				int triangleId = tex1Dfetch( texCellTriangleIds, i );

				// Check for intersection
				hitMT( triangleId, ray, hit );
		        
				// Go to next triangle
				++i;
			}

			// If found hit
			if( hit.dist < MAX_VALUE )
			{
				camera.frameBuffer[pixelAddress] = shade( ray, hit );
				return;
			}
		}

		// Get next cell
		// The above code could all go here, but it is faster to do it early
		cell = tex1Dfetch( texCellPointers, grid.to1dCoord( cellCoords ) );

	} while( cellCoords.x != outLimit.x && cellCoords.y != outLimit.y && cellCoords.z != outLimit.z );

	// Background color
	camera.frameBuffer[pixelAddress] = make_color( 255 );
}

/*

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// Triangle-ABB Overlap Axis Tests
// These were defines in original algorithm
static inline __device__ void axisTestX( const float3& boxHalfSizes, const float3& vA, const float3& vB, 
	                                     float a, float b, float fa, float fb, float& minp, float& maxp, float& rad )
{
	const float p0 = a*vA.y - b*vA.z;
	const float p2 = a*vB.y - b*vB.z;

	if( p0 < p2 )
	{
		minp=p0;
		maxp=p2;
	}
	else
	{
		minp=p2;
		maxp=p0;
	}

	rad = fa * boxHalfSizes.y + fb * boxHalfSizes.z;
}

static inline __device__ void axisTestY( const float3& boxHalfSizes, const float3& vA, const float3& vB, 
	                                     float a, float b, float fa, float fb, float& minp, float& maxp, float& rad )
{
	const float p0 = -a*vA.x + b*vA.z;
	const float p2 = -a*vB.x + b*vB.z;

	if( p0 < p2 )
	{
		minp = p0;
		maxp = p2;
	}
	else
	{
		minp = p2;
		maxp = p0;
	}

	rad = fa * boxHalfSizes.x + fb * boxHalfSizes.z;
}

static inline __device__ void axisTestZ( const float3& boxHalfSizes, const float3& vA, const float3& vB, 
	                                     float a, float b, float fa, float fb, float& minp, float& maxp, float& rad )
{
	const float p1 = a*vA.x - b*vA.y;
	const float p2 = a*vB.x - b*vB.y;

	if( p2 < p1 )
	{
		minp = p2;
		maxp = p1;
	}
	else
	{
		minp = p1;
		maxp = p2;
	}

	rad = fa * boxHalfSizes.x + fb * boxHalfSizes.y;
}

static inline __device__ void findMinMax( float a, float b, float c, float& minp, float& maxp )
{
	minp = a;
	maxp = a;

	if( b < minp )
		minp = b;
	if( b > maxp )
		maxp = b;
	if( c < minp )
		minp = c;
	if( c > maxp )
		maxp = c;
}

static inline __device__ bool planeBoxOverlap( const float3& boxHalfSizes, const float3& normal, 
	                                           const float3& vertex )
{
	float3 vmin;
	float3 vmax;

	//////////////////////////////////////////////////////
	// X
	float v = vertex.x;					// -NJMP-
	if( normal.x > 0.0f )
	{
		vmin.x = -boxHalfSizes.x - v;	// -NJMP-
		vmax.x =  boxHalfSizes.x - v;	// -NJMP-
	}
	else
	{
		vmin.x =  boxHalfSizes.x - v;	// -NJMP-
		vmax.x = -boxHalfSizes.x - v;	// -NJMP-
	}

	//////////////////////////////////////////////////////
	// Y
	v = vertex.y;					// -NJMP-
	if( normal.y > 0.0f )
	{
		vmin.y = -boxHalfSizes.y - v;	// -NJMP-
		vmax.y =  boxHalfSizes.y - v;	// -NJMP-
	}
	else
	{
		vmin.y =  boxHalfSizes.y - v;	// -NJMP-
		vmax.y = -boxHalfSizes.y - v;	// -NJMP-
	}

	//////////////////////////////////////////////////////
	// Z
	v = vertex.z;					// -NJMP-
	if( normal.z > 0.0f )
	{
		vmin.z = -boxHalfSizes.z - v;	// -NJMP-
		vmax.z =  boxHalfSizes.z - v;	// -NJMP-
	}
	else
	{
		vmin.z =  boxHalfSizes.z - v;	// -NJMP-
		vmax.z = -boxHalfSizes.z - v;	// -NJMP-
	}

	if( dot( normal, vmin ) > 0.0f ) // -NJMP-
		return false;

	if( dot( normal, vmax ) >= 0.0f ) // -NJMP-
		return true;

	return false;
}

static inline __device__ bool overlaps( float3 minv, float3 maxv, float3 v0, float3 v1, float3 v2 )
{
// Tomas Akenine-M�ller Aabb-triangle overlap test (author's optimized and fixed version 18-06-2001)
	// http://www.cs.lth.se/home/Tomas_Akenine_Moller/code/tribox3.txt

	// Use separating axis theorem to test overlap between triangle and box
	// Need to test for overlap in these cases (directions):
	// 1) the {x,y,z}-directions (actually, since we use the Aabb of the triangle we do not even need to test these)
	// 2) normal of the triangle
	// 3) crossproduct(edge from tri, {x,y,z}-direction), this gives 3x3=9 more tests

	// Compute box center and halfsizes
	float3 boxCenter = ( maxv + minv ) * 0.5f;
	float3 boxHalfSizes = ( maxv - minv ) * 0.5f;

	// This is the fastest branch on Sun
	// Move everything so that the boxcenter is in (0,0,0)
	float3 vv0 = v0 - boxCenter;
	float3 vv1 = v1 - boxCenter;
	float3 vv2 = v2 - boxCenter;

	// Compute triangle edges
	float3 e0 = vv1 - vv0;
	float3 e1 = vv2 - vv1;
	float3 e2 = vv0 - vv2;

	// Case 3: test the 9 tests first (this was faster)
	float minp;
	float maxp;
	float rad;

	// Edge 0
	float fex = abs( e0.x );
	float fey = abs( e0.y );
	float fez = abs( e0.z );

	axisTestX( boxHalfSizes, vv0, vv2, e0.z, e0.y, fez, fey, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	axisTestY( boxHalfSizes, vv0, vv2, e0.z, e0.x, fez, fex, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	axisTestZ( boxHalfSizes, vv1, vv2, e0.y, e0.x, fey, fex, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	// Edge 1
	fex = abs( e1.x );
	fey = abs( e1.y );
	fez = abs( e1.z );

	axisTestX( boxHalfSizes, vv0, vv2, e1.z, e1.y, fez, fey, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	axisTestY( boxHalfSizes, vv0, vv2, e1.z, e1.x, fez, fex, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	axisTestZ( boxHalfSizes, vv0, vv1, e1.y, e1.x, fey, fex, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	// Edge 2
	fex = abs( e2.x );
	fey = abs( e2.y );
	fez = abs( e2.z );

	axisTestX( boxHalfSizes, vv0, vv1, e2.z, e2.y, fez, fey, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	axisTestY( boxHalfSizes, vv0, vv1, e2.z, e2.x, fez, fex, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	axisTestZ( boxHalfSizes, vv1, vv2, e2.y, e2.x, fey, fex, minp, maxp, rad );
	if( minp > rad || maxp < -rad )
		return false;

	// Case 1: first test overlap in the {x,y,z}-directions
	// Find min, max of the triangle each direction, and test for overlap in that direction 
	// this is equivalent to testing a minimal Aabb around the triangle against the Aabb

	// Test in X-direction
	findMinMax( vv0.x, vv1.x, vv2.x, minp, maxp );
	if( minp > boxHalfSizes.x || maxp < -boxHalfSizes.x )
		return false;

	// Test in Y-direction
	findMinMax( vv0.y, vv1.y, vv2.y, minp, maxp );
	if( minp > boxHalfSizes.y || maxp < -boxHalfSizes.y )
		return false;

	// Test in Z-direction
	findMinMax( vv0.z, vv1.z, vv2.z, minp, maxp );
	if( minp > boxHalfSizes.z || maxp < -boxHalfSizes.z )
		return false;

	// Case 2: test if the box intersects the plane of the triangle
	// compute plane equation of triangle: normal*x+d=0
	float3 normal = cross( e0, e1 );

	// -NJMP- (line removed here)

	if( !planeBoxOverlap( boxHalfSizes, normal, vv0 ) ) // -NJMP-
		return false;

	return true;
}

__global__ void countTriangles( unsigned int vertexCount, Grid grid, float* output )
{
	//////////////////////////////////////////////////////////////////////////
	// Step 1: compute cell 1D and 3D coordinates
	//////////////////////////////////////////////////////////////////////////
	int cellIdx = blockIdx.x * blockDim.x + threadIdx.x;

	// Skip padding threads
	if( cellIdx >= grid.gridSize.x * grid.gridSize.y * grid.gridSize.z )
		return;

	float3 cellCoords = grid.to3dCoord( cellIdx );

	//////////////////////////////////////////////////////////////////////////
	// Step 2: iterate through all triangles and count how many intersect this cell
	//////////////////////////////////////////////////////////////////////////
	float3 cellMinv = grid.boxMin + cellCoords * grid.cellSize;
	float3 cellMaxv = cellMinv + grid.cellSize;

	// Output counter
	float numOverlapTriangles = 0.0f;

__shared__ float3 s_triVertices[g_blockWidth];

#ifdef __DEVICE_EMULATION__
	#define s_vertex(i) CUT_BANK_CHECKER( s_triVertices, i )
#else
	#define s_vertex(i) s_triVertices[i]
#endif

	// For each triangle, check if it overlaps current cell
	for( unsigned int v = 0; v < vertexCount; v+=g_blockWidth )
	{
		// Each thread loads a single vertex to shared memory
		s_vertex(threadIdx.x) = tex1Dfetch3( texVertices, v + threadIdx.x );

		// Sync before processing shared memory
		__syncthreads();

		// Loop over shared vertices
		for( unsigned int i = 0; i < g_blockWidth; i+=3 )
		{
			float3 v0 = s_vertex(i);
			float3 v1 = s_vertex(i+1);
			float3 v2 = s_vertex(i+2);

			if( overlaps( cellMinv, cellMaxv, v0, v1, v2 ) )
				++numOverlapTriangles;
		}

		// Sync before loading another batch of vertices to shared memory
        __syncthreads();
	}

	// For each triangle, check if it overlaps current cell
	//for( unsigned int t = 0; t < vertexCount; t+=3 )
	//{
	//	// Get triangle vertices
	//	//float3 v0 = make_float3( t+1, t, t );
	//	//float3 v1 = make_float3( t, t, t+1 );
	//	//float3 v2 = make_float3( t, t+1, t );

	//	float3 v0 = tex1Dfetch3( texVertices, t );
	//	float3 v1 = tex1Dfetch3( texVertices, t+1 );
	//	float3 v2 = tex1Dfetch3( texVertices, t+2 );

	//	//float3 triMinv = min( v0, min( v1, v2 ) );
	//	//float3 triMaxv = max( v0, max( v1, v2 ) );

	//	//float3 triCellStart = floor( clamp( grid.worldToVoxel( triMinv ), 
	//	//                                    make_float3( 0.0f ), grid.gridSize - make_float3( 1.0f ) ) );

	//	//float3 triCellEnd = floor( clamp( grid.worldToVoxel( triMaxv ), 
	//	//                                    make_float3( 0.0f ), grid.gridSize - make_float3( 1.0f ) ) );

	//	//bool overlap = ( triCellStart.x <= cellCoords.x ) && ( cellCoords.x <= triCellEnd.x ) &&
	//	//               ( triCellStart.y <= cellCoords.y ) && ( cellCoords.y <= triCellEnd.y ) &&
	//	//               ( triCellStart.z <= cellCoords.z ) && ( cellCoords.z <= triCellEnd.z );

	//	//if( overlap )
	//	//	++numOverlapTriangles;

	//	if( overlaps( cellMinv, cellMaxv, v0, v1, v2 ) )
	//	//float3 v = v0 + v1 + v2;
	//	//if( v.x > 0 && v.y > 0 && v.z > 0 )
	//		++numOverlapTriangles;
	//}

	//////////////////////////////////////////////////////////////////////////
	// Step 3: write number of overlapping triangles in output
	//////////////////////////////////////////////////////////////////////////
	output[cellIdx] = numOverlapTriangles;
}
*/
