
#include <hip/hip_runtime.h>
/*

#include "KernelDependencies.h"


// TODO: teste collecting cells first and intersecting later. reduces registers, 
// but without mem coalescing it is actually 1/3 original performance (300fps for 1 triangle)

// Kernel dependencies
#include "KernelDependencies.h"

// Global constants
static __device__ const float EPSILON   = 2e-6f;
static __device__ const float MAX_VALUE = 1e20f;

// Grid data
texture<int2, 1> texCellPointers;
texture<int, 1>  texCellTriangleIds;

// Geometry data
texture<float4, 1>  texVertices;
texture<float4, 1>  texNormals;

// 3-component texture fetch
template<class Type>
static inline __device__ float3 tex1Dfetch3( texture<Type, 1, cudaReadModeElementType> texRef, int idx )
{
	float4 texel = tex1Dfetch( texRef, idx );
	return make_float3( texel.x, texel.y, texel.z );
}

// Color conversion functions
static inline __device__ uchar4 make_color( unsigned char r, unsigned char g, unsigned char b )
{
    return make_uchar4( r, g, b, 255 );
}

static inline __device__ uchar4 make_color( unsigned char value )
{
    return make_uchar4( value, value, value, 255 );
}

static inline __device__ uchar4 make_color( float3 color, float alpha )
{
    return make_uchar4( color.x * 255.0f, color.y * 255.0f, color.z * 255.0f, alpha * 255.0f );
}

// Clip ray against bounding box
static inline __device__ bool rayHitsBoundingBox( float3 boxMin, float3 boxMax, Ray& ray )
{
	float3 t1 = ( boxMin - ray.orig ) * ray.invDir;
	float3 t2 = ( boxMax - ray.orig ) * ray.invDir;
	
	float3 minT1T2 = min( t1, t2 );
	float3 maxT1T2 = max( t1, t2 );
	
	ray.tnear = max( max( minT1T2.x, minT1T2.y ), minT1T2.z );
	ray.tfar = min( min( maxT1T2.x, maxT1T2.y ), maxT1T2.z );

	return ray.tnear <= ray.tfar;
}

// Compute intersection between triangle and current ray
// Moller-Trumbore algorithm
static inline __device__ void hitMT( int triangleId, const Ray& ray, Hit& hit )
{
	float3 v0 = tex1Dfetch3( texVertices, triangleId );
	float3 v1 = tex1Dfetch3( texVertices, triangleId + 1 );
	float3 v2 = tex1Dfetch3( texVertices, triangleId + 2 );

	float3 e1 = v1 - v0;
	float3 e2 = v2 - v0;
	float3 tvec = ray.orig - v0;

	float3 p = cross(ray.dir, e2);
	float3 q = cross(tvec, e1);

	float invdet = 1.0f / dot(p, e1);

	float u = dot(p, tvec) * invdet;
	float v = dot(q, ray.dir) * invdet;

	// Update hit
	bool isHit = (u >= 0.0f) && (v >= 0.0f) && (u + v <= 1.0f);
	
	float t = dot(q, e2) * invdet;

	// Update hit
	isHit &= (t > 0.0f) && (t < ray.tfar + EPSILON) && (t < hit.dist);

	if( isHit )
	{
		hit.id = triangleId;
		hit.u = u;
		hit.v = v;
		hit.dist = t;
	}
}

// Compute interpolated shading normal
static inline __device__ float3 computeShadingNormal( const Hit& hit )
{
	// Get triangle normals
    float3 n0 = tex1Dfetch3( texNormals, hit.id );
    float3 n1 = tex1Dfetch3( texNormals, hit.id + 1 );
    float3 n2 = tex1Dfetch3( texNormals, hit.id + 2 );

	return normalize( n0 * ( 1.0f - ( hit.u + hit.v ) ) +	// v0 coord
		              n1 * hit.u +							// v1 coord
					  n2 * hit.v );							// v2 coord
}

static inline __device__ uchar4 shade( const Ray& ray, const Hit& hit )
{
	// Need normalized ray direction
	float3 rayDirNormalized = normalize( ray.dir );

	// Compute interpolated shading normal
	float3 sampleNormal = computeShadingNormal( hit );

	// Hard-coded material information
	const float3 ambient = make_float3( 0.1f, 0.1f, 0.1f );
	const float3 diffuse = make_float3( 0.0f, 0.0f, 1.0f );

	// Headlight illumination
	float nDotD = -dot( sampleNormal, rayDirNormalized );
	float3 sampleColor = ( ambient + ( diffuse - ambient ) * nDotD ) * diffuse;
	return make_color( sampleColor, 1.0f );
}

static inline __device__ int2 getCell( const Grid& grid, float3 coords )
{
	int linearCoord = coords.x + coords.y*grid.gridSize.x + coords.z*grid.gridSize.x*grid.gridSize.y;
	return tex1Dfetch( texCellPointers, linearCoord );
}

static inline __device__ bool isEmpty( int2 cell )
{
	return cell.y == 0;
}

static inline __device__ int getTriangleStart( int2 cell )
{
	return cell.x;
}

static inline __device__ int getTriangleCount( int2 cell )
{
	return cell.y;
}

// Kernel for entire ray tracing pipeline
// Each thread traces a single ray
// Each thread computes its own ray direction
__global__ void rayTrace( Camera camera, Grid grid )
{
	//////////////////////////////////////////////////////////////////////////
	// Step 1: compute my screen coordinates
	//////////////////////////////////////////////////////////////////////////
	unsigned int screenX = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int screenY = blockIdx.y * blockDim.y + threadIdx.y;

	float uStep = (float)screenX * camera.invScreenWidth;
	float vStep = (float)screenY * camera.invScreenHeight;

	unsigned int pixelAddress = screenY*camera.screenWidth + screenX;

	//////////////////////////////////////////////////////////////////////////
	// Step 2.1: minimal ray attributes for box clipping
	//////////////////////////////////////////////////////////////////////////
	Ray ray;

	ray.orig = camera.position;

	ray.dir = camera.baseDir + camera.nearU*uStep + camera.nearV*vStep;
	ray.invDir = rcp( ray.dir );

    // If don't hit bbox in local space, no need to trace underlying triangles
	if( !rayHitsBoundingBox( grid.boxMin, grid.boxMax, ray ) )
	{
		// Background color
		camera.frameBuffer[pixelAddress] = make_color( 255 );
		return;
	}

	//////////////////////////////////////////////////////////////////////////
	// Step 2.2: remaining ray attributes
	//////////////////////////////////////////////////////////////////////////
	float3 dirSignBits;
    dirSignBits.x = ( ray.dir.x < 0.0f )? 1.0f : 0.0f;
    dirSignBits.y = ( ray.dir.y < 0.0f )? 1.0f : 0.0f;
    dirSignBits.z = ( ray.dir.z < 0.0f )? 1.0f : 0.0f;

	float3 notDirSignBits = not( dirSignBits );

	//////////////////////////////////////////////////////////////////////////
	// Initial setup                                                        
	//////////////////////////////////////////////////////////////////////////
	// 1. Find initial cell where ray begins

	// Since ray was already clipped against bbox (grid),
	// ray tnear gives us the starting t (thus the start point as well)
	float3 startPoint = ray.orig + ray.dir * ray.tnear;
	
	// Floor is needed when working with float values (equivalent to truncating to int)
	float3 cellCoords = floor( clamp( grid.worldToVoxel( startPoint ), 
		                              make_float3( 0.0f ), grid.gridSize - make_float3( 1.0f ) ) );

	// 2. Compute stepX, stepY, stepZ
	float3 cellStep = -dirSignBits + notDirSignBits;

	// 3 Compute out of grid limits
	float3 outLimit = -dirSignBits + grid.gridSize * notDirSignBits;

	// 4. Compute tDeltaX, tDeltaY, tDeltaZ
	float3 tDelta = abs( grid.cellSize * ray.invDir );

	// 5. Compute tNextX, tNextY, tNextZ
	float3 tMax = ( grid.voxelToWorld( cellCoords + notDirSignBits ) - ray.orig ) * ray.invDir;

	//////////////////////////////////////////////////////////////////////////
	// Trace ray through grid                                               
	//////////////////////////////////////////////////////////////////////////
	// Find first non-empty cell
	int2 cell = getCell( grid, cellCoords );

	// Store hit information
	Hit hit;
	hit.dist = MAX_VALUE;

	// Minimum tMax in all 3 dimensions, used for logical comparison to determine next cell
	float minTmax;

	// Stores 1 for next cell dimension and 0 for the others, used to select next cell
	float3 comp;

	// While inside grid
	do
	{
		// Early traversal pre-computation
		// Already begin computing next cell to be visited before testing current one
		// To go to next cell, need to decide which dimension is next
		// comp stores 1 for next dimension and 0 for others
 		minTmax = min( min( tMax.x, tMax.y ), tMax.z );
 		comp = step( tMax - make_float3( minTmax ), make_float3( EPSILON ) );

		// Step ray according to comp
 		cellCoords += cellStep * comp;
 		tMax += tDelta * comp;

		// If cell contains triangles, test intersection
		if( !isEmpty( cell ) )
		{
			// We send the smallest tMax as the maximum valid distance
			// This avoids false intersections outside current cell
			ray.tfar = minTmax;

			// Iterate through triangles in given cell and compute nearest intersection, if any
			int i = getTriangleStart( cell );
			int end = getTriangleStart( cell ) + getTriangleCount( cell );

			while( i < end )
			{
				// Get triangle id
				int triangleId = tex1Dfetch( texCellTriangleIds, i );

				// Check for intersection
				hitMT( triangleId, ray, hit );
		        
				// Go to next triangle
				++i;
			}

			// If found hit
			if( hit.dist < MAX_VALUE )
			{
				camera.frameBuffer[pixelAddress] = shade( ray, hit );
				return;
			}
		}

		// Get next cell
		// The above code could all go here, but it is faster to do it early
		cell = getCell( grid, cellCoords );

	} while( cellCoords.x != outLimit.x && cellCoords.y != outLimit.y && cellCoords.z != outLimit.z );

	// Background color
	camera.frameBuffer[pixelAddress] = make_color( 255 );
}

//////////////////////////////////////////////////////////////////////

// TODO: need methods from "RayTracing_kernel.cu"
// TODO: all goes well until traceValidRays kernel -> requires 40+ registers!!!

__global__ void initRays( Camera camera, float4* rayOrigins, float4* rayDirs, float4* rayInvDirs )
{
	//////////////////////////////////////////////////////////////////////////
	// Step 1: compute my screen coordinates
	//////////////////////////////////////////////////////////////////////////
	unsigned int screenX = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int screenY = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int pixelAddress = screenY*camera.screenWidth + screenX;

	//////////////////////////////////////////////////////////////////////////
	// Step 2: compute ray directions
	//////////////////////////////////////////////////////////////////////////
	float uStep = (float)screenX * camera.invScreenWidth;
	float vStep = (float)screenY * camera.invScreenHeight;
	float3 rayDir = camera.baseDir + camera.nearU*uStep + camera.nearV*vStep;

	//////////////////////////////////////////////////////////////////////////
	// Step 3: store values for next kernel
	//////////////////////////////////////////////////////////////////////////
	rayOrigins[pixelAddress] = make_float4( camera.position );
	rayDirs[pixelAddress] = make_float4( rayDir );
	rayInvDirs[pixelAddress] = make_float4( rcp( rayDir ) );
}

__global__ void hitSceneBox( Camera camera, Grid grid, 
							 float4* rayOrigins, float4* rayInvDirs, float* tnears )
{
	//////////////////////////////////////////////////////////////////////////
	// Step 1: compute my screen coordinates
	//////////////////////////////////////////////////////////////////////////
	unsigned int screenX = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int screenY = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int pixelAddress = screenY*camera.screenWidth + screenX;

	//////////////////////////////////////////////////////////////////////////
	// Step 2: minimal ray attributes for box clipping
	//////////////////////////////////////////////////////////////////////////
	float3 rayOrig   = make_float3( rayOrigins[pixelAddress] );
	float3 rayInvDir = make_float3( rayInvDirs[pixelAddress] );

	//////////////////////////////////////////////////////////////////////////
	// Step 3: clip ray against grid bounding box
	//////////////////////////////////////////////////////////////////////////
	float3 t1 = ( grid.boxMin - rayOrig ) * rayInvDir;
	float3 t2 = ( grid.boxMax - rayOrig ) * rayInvDir;
	
	float3 minT1T2 = min( t1, t2 );
	float3 maxT1T2 = max( t1, t2 );
	
	float tnear = max( max( minT1T2.x, minT1T2.y ), minT1T2.z );
	float tfar = min( min( maxT1T2.x, maxT1T2.y ), maxT1T2.z );

	bool hit = tnear <= tfar;

	//////////////////////////////////////////////////////////////////////////
	// Step 4: store values for next kernel
	//////////////////////////////////////////////////////////////////////////
	tnears[pixelAddress] = ( hit )? tnear : MAX_VALUE;
}

__global__ void traceValidRays( Camera camera, Grid grid, 
							    float4* rayOrigins, float4* rayDirs, float4* rayInvDirs, float* tnears, Hit* hits )
{
	//////////////////////////////////////////////////////////////////////////
	// Step 1: compute my screen coordinates
	//////////////////////////////////////////////////////////////////////////
	unsigned int screenX = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int screenY = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int pixelAddress = screenY*camera.screenWidth + screenX;

	//////////////////////////////////////////////////////////////////////////
	// Step 2: abort invalid rays
	//////////////////////////////////////////////////////////////////////////
	Ray ray;
	ray.tnear = tnears[pixelAddress];

	if( ray.tnear == MAX_VALUE )
	{
		// Background color
		camera.frameBuffer[pixelAddress] = make_color( 255 );
		return;
	}

	//////////////////////////////////////////////////////////////////////////
	// Step 3: remaining ray attributes
	//////////////////////////////////////////////////////////////////////////
	ray.orig   = make_float3( rayOrigins[pixelAddress] );
	ray.dir    = make_float3( rayDirs[pixelAddress] );
	ray.invDir = make_float3( rayInvDirs[pixelAddress] );

	float3 dirSignBits;
    dirSignBits.x = ( ray.dir.x < 0.0f )? 1.0f : 0.0f;
    dirSignBits.y = ( ray.dir.y < 0.0f )? 1.0f : 0.0f;
    dirSignBits.z = ( ray.dir.z < 0.0f )? 1.0f : 0.0f;

	float3 notDirSignBits = not( dirSignBits );

	//////////////////////////////////////////////////////////////////////////
	// Initial setup                                                        
	//////////////////////////////////////////////////////////////////////////
	// 1. Find initial cell where ray begins

	// Since ray was already clipped against bbox (grid),
	// ray tnear gives us the starting t (thus the start point as well)
	float3 startPoint = ray.orig + ray.dir * ray.tnear;
	
	// Floor is needed when working with float values (equivalent to truncating to int)
	float3 cellCoords = floor( clamp( grid.worldToVoxel( startPoint ), 
		                              make_float3( 0.0f ), grid.cellTotal - make_float3( 1.0f ) ) );

	// 2. Compute stepX, stepY, stepZ
	float3 cellStep = -dirSignBits + notDirSignBits;

	// 3 Compute out of grid limits
	float3 outLimit = -dirSignBits + grid.cellTotal * notDirSignBits;

	// 4. Compute tDeltaX, tDeltaY, tDeltaZ
	float3 tDelta = abs( grid.cellSize * ray.invDir );

	// 5. Compute tNextX, tNextY, tNextZ
	float3 tMax = ( grid.voxelToWorld( cellCoords + notDirSignBits ) - ray.orig ) * ray.invDir;

	//////////////////////////////////////////////////////////////////////////
	// Trace ray through grid                                               
	//////////////////////////////////////////////////////////////////////////
	// Find first non-empty cell
	int2 cell = getCell( grid, cellCoords );

	// Store hit information
	Hit hit;
	hit.dist = MAX_VALUE;

	// Minimum tMax in all 3 dimensions, used for logical comparison to determine next cell
	float minTmax;

	// Stores 1 for next cell dimension and 0 for the others, used to select next cell
	float3 comp;

	// While inside grid
	do
	{
		// Early traversal pre-computation
		// Already begin computing next cell to be visited before testing current one
		// To go to next cell, need to decide which dimension is next
		// comp stores 1 for next dimension and 0 for others
 		minTmax = min( min( tMax.x, tMax.y ), tMax.z );
 		comp = step( tMax - make_float3( minTmax ), make_float3( EPSILON ) );

		// Step ray according to comp
 		cellCoords += cellStep * comp;
 		tMax += tDelta * comp;

		// If cell contains triangles, test intersection
		if( !isEmpty( cell ) )
		{
			// We send the smallest tMax as the maximum valid distance
			// This avoids false intersections outside current cell
			ray.tfar = minTmax;

			// Iterate through triangles in given cell and compute nearest intersection, if any
			int i = getTriangleStart( cell );
			int end = getTriangleStart( cell ) + getTriangleCount( cell );

			while( i < end )
			{
				// Get triangle id
				int triangleId = tex1Dfetch( texCellTriangleIds, i );

				// Check for intersection
				hitMT( triangleId, ray, hit );
		        
				// Go to next triangle
				++i;
			}

			// If found hit
			if( hit.dist < MAX_VALUE )
				break;
		}

		// Get next cell
		// The above code could all go here, but it is faster to do it early
		cell = getCell( grid, cellCoords );

	} while( cellCoords.x != outLimit.x && cellCoords.y != outLimit.y && cellCoords.z != outLimit.z );

	//////////////////////////////////////////////////////////////////////////
	// Step 4: store values for next kernel
	//////////////////////////////////////////////////////////////////////////
	hits[pixelAddress] = hit;
}

__global__ void shadeHits( Camera camera, float4* rayDirs, Hit* hits )
{
	//////////////////////////////////////////////////////////////////////////
	// Step 1: compute my screen coordinates
	//////////////////////////////////////////////////////////////////////////
	unsigned int screenX = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int screenY = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int pixelAddress = screenY*camera.screenWidth + screenX;

	//////////////////////////////////////////////////////////////////////////
	// Step 2: abort rays that hit nothing
	//////////////////////////////////////////////////////////////////////////
	Hit hit = hits[pixelAddress];

	if( hit.dist == MAX_VALUE )
	{
		camera.frameBuffer[pixelAddress] = make_color( 255 );
		return;
	}

	//////////////////////////////////////////////////////////////////////////
	// Step 3: shade intersected rays
	//////////////////////////////////////////////////////////////////////////
	Ray ray;
	ray.dir = make_float3( rayDirs[pixelAddress] );

	camera.frameBuffer[pixelAddress] = shade( ray, hit );
}

*/
